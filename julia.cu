/*H*
 * FILENAME: julia.cu
 *
 * AUTHOR: Andrew McKenzie
 * UNE EMAIL: amcken33@myune.edu.au
 * STUDENT NUMBER: 220263507
 *
 * PURPOSE: Generate a bitmap image of a fractal representation of the
 * Julia set.
 *
 * This program takes in a width and height for the image. It then uses
 * CUDA C to calculate each pixel's RGB values from the julia algorithm and
 * once they are returned to the host the program sets the pixels in the
 * bitmap and saves the file.
 *
 * COMPILING: The included makefile can be run with the 'make' command.
 *
 * RUNNING: The program is run by the following:
 *      ./julia <width> <height>
 *
 * Run Target Example: make run <width> <height>
 *
 * Author Recommendation: run with width 2160 and height 3240
 * (This image is now my desktop background)
 *
 * As per the Linux Kernel C programming guide:
 * - Function names use snake case for emphasis.
 * - Variables use camel case for brevity.
 * - Global variables use snake case.
 * - Constants and macros use snake case and are upper case.
 * - Everything except function declarations use K&R style braces.
 * - Functions use Allman style braces.
 *
 *H*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "bmpfile.h"
#include "julia_utils.h"
#include "julia_function_utils.cuh"
#include "macros.h"

#define FILENAME "my_julia_fractal.bmp"


/**
 * main() - Main function for the julia program.
 * @arg1: Number of args from the terminal.
 * @arg2: Array of the args from the terminal.
 *
 * The function follows these generic steps:
 * - It sets up device and host arrays for the pixels
 * - Calculates the pixels from the given function
 * - Sets the pixels in the bitmap and saves the file
 *
 * Reference: This program is heavily reliant on the provided
 * julia.c program and the examples used in COSC330 lectures.
 *
 * Return: Int, zero on success, non-zero on failure.
 */
int main(int argc, char **argv) {
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;

        int width, height;
        parse_args(argc, argv, &width, &height);

        // Bitmap with 32 bits for each pixel (RGBA)
        bmpfile_t *bmp = bmp_create(width, height, 32);

        // Offset for the Julia image in the bitmap image
        int xoffset = -(width - 1) / 2;
        int yoffset = (height - 1) / 2;

        // Size of image
        size_t size = width * height * RGB_LENGTH * sizeof(float);

        // Pointer to memory for hosts pixels
        float* h_pixels = (float*)malloc(size);
        if (h_result == NULL)
        {
                fprintf(stderr, "Failed to allocate host vectors!\n");
                exit(EXIT_FAILURE);
        }

        // Pointer to devices pixels
        float* d_pixels;
        err = hipMalloc((void**)&d_pixels, size);
        checkError(err, "Failed to allocate memory for device");

        //
        dim3 threadsPerBlock(8, 8);
        dim3 blocksPerGrid((width + threadsPerBlock.x - 1) /
                threadsPerBlock.x, (height + threadsPerBlock.y - 1)
                        / threadsPerBlock.y);

        // TODO Remove this
        printf("Launching kernel with %d blocks and %d threads per block\n",
               blocksPerGrid.x * blocksPerGrid.y,
               threadsPerBlock.x * threadsPerBlock.y);

        // Kernel call
        juliaSetKernel<<<blocksPerGrid, threadsPerBlock>>>(d_pixels,
                                                           width,
                                                           height,
                                                           xoffset,
                                                           yoffset);

        err = hipGetLastError();
        checkError(err, "Failed in call to juliaSetKernel");

        // Copying results back to host
        err = hipMemcpy(h_pixels, d_pixels, size, hipMemcpyDeviceToHost);
        checkError(err, "Failed to copy d_result to Host");

        set_pixels(height, width, h_pixels, bmp);

        // Attempting to save new file
        if (bmp_save(bmp, FILENAME) == 0) {
                fprintf(stderr, "Failed to save bmp file (error code %s)!\n",
                        hipGetErrorString(err));
        }

        // Freeing up resources
        bmp_destroy(bmp);
        free(h_pixels);
        err = hipFree(d_pixels);
        checkError(err, "Failed to free device memory");

        // Deinitialising for good practice
        err = hipDeviceReset();
        checkError(err, "Failed to deinitialise the device");

        return 0;
}
